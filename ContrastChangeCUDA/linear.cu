#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "linear.cuh"

__global__ void linearKernel(int* color, int width, int height)
{

}


void* linearContrast(int * color, int Width, int Height) {
	int* dev_col;
	int n = 32;
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	hipMalloc((void **)&dev_col, sizeof(int)*Height*Width * 5);
	hipMemcpy(dev_col, color, sizeof(int)*Height*Width * 5, hipMemcpyHostToDevice);
	dim3 grid((Width + prop.maxThreadsPerBlock / n - 1) / (prop.maxThreadsPerBlock / n), (Height + prop.maxThreadsPerBlock / n - 1) / (prop.maxThreadsPerBlock / n), 1);
	dim3 threads(prop.maxThreadsPerBlock / n, prop.maxThreadsPerBlock / n, 1);
	linearKernel << <grid, threads >> > (dev_col, Width, Height);
	hipMemcpy(color, dev_col, sizeof(int)*Height*Width * 5, hipMemcpyDeviceToHost);
	hipFree(dev_col);
	return(color);
}
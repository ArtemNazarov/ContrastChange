#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "equalization.cuh"

#define H_SIZE 256


//__global__ void frKernel(double* color)
//{
//
//}

__global__ void equalizeKernel(double* color, int width, int height) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int jdy = blockIdx.y*blockDim.y + threadIdx.y;

	if (idx == 0 || jdy == 0 || idx == width - 1 || jdy == height - 1)
		color[idx*width + jdy] = color[idx*width + jdy];
	else {
		int col = 0;
		color[idx * width + jdy] /= width * height;
		color[idx * width + jdy] /= color[idx * width + jdy] + color[(idx - 1) * width + jdy];
	}
}

void* equalizeHist(double * color, int Width, int Height) {
	double* dev_color;
	int n = 32;
	hipDeviceProp_t prop;
	size_t size = sizeof(double) * Width * Height;
	hipGetDeviceProperties(&prop, 0);
	hipError_t result = hipMalloc((void **)&dev_color, size);
	if (result != hipError_t::hipSuccess) {
		return NULL;
	}
	result = hipMemcpy(dev_color, color, size, hipMemcpyHostToDevice);
	if (result != hipError_t::hipSuccess) {
		return NULL;
	}
	dim3 grid((Width + prop.maxThreadsPerBlock / n - 1) / (prop.maxThreadsPerBlock / n), (Height + prop.maxThreadsPerBlock / n - 1) / (prop.maxThreadsPerBlock / n), 1);
	dim3 threads(prop.maxThreadsPerBlock / n, prop.maxThreadsPerBlock / n, 1);
	//dim3 threads(128, 128);
	//dim3 blocks((Width + threads.x - 1) / threads.x, (Height + threads.y - 1) / threads.y);
	equalizeKernel << <grid, threads >> > (dev_color, Width, Height);
	hipMemcpy(color, dev_color, size, hipMemcpyDeviceToHost);
	hipFree(dev_color);
	return(color);
}

void calclc(int* erg) {

}

int calculate(double* erg) {
	double * dev_srcImage;
	size_t size = sizeof(double) * 2 * 4;

	if (hipMalloc((void**)&dev_srcImage, size) != hipError_t::hipSuccess) return -1;
	// ������������ ���������� ����� �� ���� ����� ����������
	dim3 threads(128, 128);
	dim3 blocks((threads.x - 1) / threads.x, (threads.y - 1) /
		threads.y);
	//kernel << <threads, blocks >> > (dev_srcImage, width, height);
	hipError_t error = hipMemcpy(erg, dev_srcImage, size,
		hipMemcpyDeviceToHost);
	if (error != hipError_t::hipSuccess) return -1;
	hipFree(dev_srcImage);
	return 0;
}